#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>




#define N 50000
#define SMOOTH 4 
#define BLOCK_SIZE 1024

/**
* timeDiff
*
* Computes the difference (in ns) between the start and end time
*/

double timeDiff(struct timespec tStart, struct timespec tEnd)
{
	struct timespec diff;

	diff.tv_sec = tEnd.tv_sec - tStart.tv_sec - (tEnd.tv_nsec<tStart.tv_nsec ? 1 : 0);
	diff.tv_nsec = tEnd.tv_nsec - tStart.tv_nsec + (tEnd.tv_nsec<tStart.tv_nsec ? 1000000000 : 0);

	return ((double)diff.tv_sec) + ((double)diff.tv_nsec) / 1e9;
}

float randn()
{
	float x1, x2, w, y1;
	do
	{
		x1 = (float)(2.0 * rand() / RAND_MAX - 1.0);
		x2 = (float)(2.0 * rand() / RAND_MAX - 1.0);
		w = x1 * x1 + x2 * x2;
	} while (w >= 1.0);

	w = (float)sqrt((-2.0 * log(w)) / w);
	y1 = x1 * w;
	return y1;
}

float function(float b, float c) {
	return (float)(sinf(0.02 *b) + sinf(0.001*b) + 0.1*c);
}


__global__ void funtion_smooth(float *x, float *y, float *yest, int n){
	int i = blockIdx.x* blockDim.x + threadIdx.x;
	float sumA=0.0, sumB=0.0;

	
	if (i < n){
		for (int j = 0; j < n ;j++){
			sumA = sumA + ((expf(-powf((x[i] - x[j]), 2) / (2 * powf(SMOOTH, 2)))) * y[j]);
			sumB = sumB + (expf(-powf((x[i] - x[j]), 2) / (2 * powf(SMOOTH, 2))));
		}
	yest[i] = sumA / sumB;
	}

	

}
int main() {

	float *x, *y, *yest_cpu,*yest_gpu, *randomArray;
	float *d_x, *d_y, *d_yest;
	float sumA, sumB;
	char outputFilenamey[] = "y.txt";
	char outputFilenameyestcpu[] = "yest_c.txt";
	char outputFilenameyestgpu[] = "yest_g.txt";
	FILE *ofpy, *ofpyestcpu, *ofpyestgpu;

	struct timespec timeVect[7];
	double timeCPU, timeGPU[7];

	x = (float *)malloc(N*sizeof(float));
	y = (float *)malloc(N*sizeof(float));
	yest_gpu = (float *)malloc(N*sizeof(float));
	yest_cpu = (float *)malloc(N*sizeof(float));
	randomArray = (float *)malloc(N*sizeof(float));


	for (int i = 0; i < N; ++i) {
		x[i] = (float)i / 10;
		randomArray[i] = randn();
		y[i] = function((float)x[i], (float)randomArray[i]);
	}

	/*Corre no cpu*/
	printf("Performing the computation on the CPU...\n");
	clock_gettime(CLOCK_REALTIME, &timeVect[0]);
	
	for (int i = 0; i < N; ++i) { //percorrer o yest
		sumA = 0;
		for (int j = 0; j < N; ++j) { //percorer o input dataset
			sumA = sumA + ((expf(-powf((x[i] - x[j]), 2) / (2 * powf(SMOOTH, 2)))) * y[j]);
		}
		sumB = 0;
		for (int j = 0; j < N; ++j)	{ //percorer o input dataset
			sumB = sumB + expf(-powf((x[i] - x[j]), 2) / (2 * powf(SMOOTH, 2)));
		}
		yest_cpu[i] = sumA / sumB;
	}
	clock_gettime(CLOCK_REALTIME, &timeVect[1]);
	timeCPU = timeDiff(timeVect[0], timeVect[1]);
	printf("    ... execution took %.6f seconds\n", timeCPU);

	printf("----------------------------------------------------------------------------\n");
	printf("Performing the computation on the GPU...\n");
	
	clock_gettime(CLOCK_REALTIME, &timeVect[0]);
	hipFree(0);
	clock_gettime(CLOCK_REALTIME, &timeVect[1]);


	hipMalloc(&d_x, N*sizeof(float));
	hipMalloc(&d_y, N*sizeof(float));
	hipMalloc(&d_yest, N *sizeof(float));
	clock_gettime(CLOCK_REALTIME, &timeVect[2]);
	
	hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);
	clock_gettime(CLOCK_REALTIME, &timeVect[3]);

	dim3 dimBlock(BLOCK_SIZE, 1, 1);
	dim3 dimGrid(N / BLOCK_SIZE + 1, 1, 1);

	funtion_smooth <<< dimGrid, dimBlock >>>(d_x, d_y, d_yest, N);
	clock_gettime(CLOCK_REALTIME, &timeVect[4]);

	hipMemcpy(yest_gpu, d_yest, N*sizeof(float), hipMemcpyDeviceToHost);
	clock_gettime(CLOCK_REALTIME, &timeVect[5]);
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_yest);
	clock_gettime(CLOCK_REALTIME, &timeVect[6]);
	
	timeGPU[0] = timeDiff(timeVect[0], timeVect[1]);
	timeGPU[1] = timeDiff(timeVect[1], timeVect[2]);
	timeGPU[2] = timeDiff(timeVect[2], timeVect[3]);
	timeGPU[3] = timeDiff(timeVect[3], timeVect[4]);
	timeGPU[4] = timeDiff(timeVect[4], timeVect[5]);
	timeGPU[5] = timeDiff(timeVect[5], timeVect[6]);
	timeGPU[6] = timeDiff(timeVect[0], timeVect[6]);
	printf("    ... execution took %.6f seconds (speedup=%.3f), corresponging to:\n", timeGPU[6], timeCPU / timeGPU[6]);
	printf("          - first call to the device           -> %.6f seconds\n", timeGPU[0]);
	printf("          - allocation of memory on the device -> %.6f seconds\n", timeGPU[1]);
	printf("          - copying data from host to device   -> %.6f seconds\n", timeGPU[2]);
	printf("          - kernel execution on the device     -> %.6f seconds\n", timeGPU[3]);
	printf("          - copying data from device to host   -> %.6f seconds\n", timeGPU[4]);
	printf("          - freeing data on the device         -> %.6f seconds\n", timeGPU[5]);
	printf("----------------------------------------------------------------------------\n");

	ofpyestcpu = fopen(outputFilenameyestcpu, "w");
	ofpy = fopen(outputFilenamey, "w");
	ofpyestgpu = fopen(outputFilenameyestgpu, "w");

	for (int j = 0; j < N; ++j){
		fprintf(ofpy, "%f\n", y[j]);
		fprintf(ofpyestcpu, "%f\n", yest_cpu[j]);
		fprintf(ofpyestgpu, "%f\n", yest_gpu[j]);
	}

	fclose(ofpyestcpu);
	fclose(ofpyestgpu);
	fclose(ofpy);
	free(x);
	free(y);
	free(yest_cpu);
	free(yest_gpu);
	free(randomArray);
	


	return 0;
}